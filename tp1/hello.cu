#include <cstdio>
#include "hip/hip_runtime.h"

__global__ void cudaHello(){
  // Afficher le message Hello World ainsi que blockidx et threadidx depuis chaque thread
  printf("Hello world from: blokidx %d threadidx %d \n", blockIdx.x, threadIdx.x);
}

int main() {
  int numBlocks = 64;
  int blockSize = 1;
  // Experimenter avec de differents blockSize (nombre de threads par block) pour les puissances de 2
  // tout en gardant le nombre total de threads egale a 64
  // A FAIRE ...
  cudaHello<<<numBlocks, blockSize>>>(); 

  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess)
    printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
  return 0;
  }
