#include "hip/hip_runtime.h"
#include <cstdio>    
#include "hip/hip_runtime.h"    

#define N 1024                     
float A[N];    
float c = 2.0;    

__device__ float dA[N];    

__global__ void multiplyArray(int n, float c)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i <  n)
  dA[i] *= c;
}

int main(int argc, char **argv)
{
  // Initialisation
  for (int i = 0; i < N; i++) { A[i] = i; }
  // Copier le tableau vers le GPU
  hipMemcpyToSymbol(HIP_SYMBOL(dA), A, N * sizeof(float), 0,
      hipMemcpyHostToDevice);
  int blockSize = 128;
  int numBlocks = N / blockSize;
  if (N % blockSize) numBlocks++;
  multiplyArray<<<(numBlocks, blockSize>>>(n, c); 
  // Recopier le tableau multiplie vers le CPU
  hipMemcpyFromSymbol(A, HIP_SYMBOL(dA), N * sizeof(float), 0,
      hipMemcpyDeviceToHost);
  printf("%lf\n", A[2]);
  return 0;
}
