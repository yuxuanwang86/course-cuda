#include <cstdio>
#include <iostream>
#include "hip/hip_runtime.h"

using namespace std;

#define N 1024

// Definir un tableau de float Ad[N] de taille statique sur le GPU
__device__ float dA[N];

int main() {
  float A[N], B[N];
  int i;
  // Initialisation
  for (i = 0; i < N; i++) { A[i] = (float)i; }

  // cudaMemcpy de A[N] vers Ad[N]
  hipMemcpyToSymbol(HIP_SYMBOL(dA), A, sizeof(float) * N, 0, hipMemcpyHostToDevice);

  // cudaMemcpy de Ad[N] vers B[N]
  hipMemcpyFromSymbol(B, HIP_SYMBOL(dA), sizeof(float) * N, 0, hipMemcpyDeviceToHost); 
  // Attendre que les kernels GPUs terminent
  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess) {
    printf("L'execution du kernel a echoue avec le code d'erreur \"%s\".\n", hipGetErrorString(cudaerr));
  }

  // Verifier le resultat
  for (i = 0; i < N; i++) { if (A[i] != B[i]) { break; } }
  if (i < N) { cout << "La copie est incorrecte!\n"; }
  else { cout << "La copie est correcte!\n"; }

  return 0;
}
