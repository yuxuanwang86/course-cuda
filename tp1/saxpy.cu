#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;


// Calcul de saxpy en utilisant 1 thread par bloc, 1 operation par thread
__global__
void saxpyBlocs(const int N, float a, const float* x, float* y)
{
  int idx;
  idx = blockIdx.x;
  if (idx < N) y[idx] = a * x[idx] + y[idx];
}


// Calcul de saxpy en utilisant blockSize threads par bloc, 1 operation par thread
__global__
void saxpyBlocsThreads(const int N, float a, const float* x, float* y)
{
  int idx;
  idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) y[idx] = a * x[idx] + y[idx];
}


// Calcul de saxpy en utilisant blockSize threads par bloc et effectuant k operation par thread dans un bloc
__global__
void saxpyBlocsThreadsKops(const int N, float a, const float* x, float* y, const int k)
{
  int idx;
  for (idx = blockIdx.x * blockDim.x * k + threadIdx.x; idx < blockIdx.x * blockDim.x * k + threadIdx.x + k; ++idx) {

    if (idx >= N) break;
    y[idx] = x[idx] * a + y[idx];
  }
}

// Fonction CPU de reference pour l'operation saxpy
void saxpy(const int N, float a, float* x, float* y)
{
  for (int i = 0; i < N; i++) { y[i] = a * x[i] + y[i]; }
}

// Verifier si le resultat dans res[N] correspond a saxpy(N, a, x, y)
void verifySaxpy(float a, float* x, float* y, float* res, int N)
{
  int i;
  for (i = 0; i < N; i++) {
    float temp = a * x[i] + y[i];
    if (std::abs(res[i] - temp) / std::max(1e-6f, temp) > 1e-6) {
      cout << res[i] << " " << temp << endl;
      break;
    }
  }
  if (i == N) {
    cout << "saxpy on GPU is correct." << endl;
  }
  else {
    cout << "saxpy on GPU is incorrect on element " << i << "." << endl;
  }
}


int main(int argc, char** argv)
{
  int blockSize;
  int k;
  float* x, * y, * res, * dx, * dy;
  float a = 2.0f;

  int N;

  if (argc < 2) {
    printf("Utilisation: ./saxpy N\n");
    return 0;
  }
  N = atoi(argv[1]);

  // Allouer et initialiser les vecteurs x, y et res sur le CPU
  x = (float*)malloc(N * sizeof(float));
  y = (float*)malloc(N * sizeof(float));
  res = (float*)malloc(N * sizeof(float));
  for (int i = 0; i < N; i++) {
    x[i] = i;
    y[i] = 1.0f;
  }

  // Allouer les vecteurs dx[N] et dy[N] sur le GPU, puis copier x et y dans dx et dy.
  hipMalloc(&dx, sizeof(float) * N);
  hipMalloc(&dy, sizeof(float) * N);
  hipMemcpy(dx, x, sizeof(float) * N, hipMemcpyHostToDevice);
  hipMemcpy(dy, y, sizeof(float) * N, hipMemcpyHostToDevice);
  // Lancer le kernel saxpyBlocs avec un nombre de bloc approprie
  saxpyBlocs << <N, 1 >> > (N, a, dx, dy);
  // Copier dy[N] dans res[N] pour la verification sur CPU
  hipMemcpy(res, dy, sizeof(float) * N, hipMemcpyDeviceToHost);