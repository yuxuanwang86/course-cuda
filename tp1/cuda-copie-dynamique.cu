#include <cstdio>
#include <iostream>
#include "hip/hip_runtime.h"

using namespace std;

int main(int argc, char **argv) {
  float *A, *B, *Ad;
  int N, i;

  if (argc < 2) {
    printf("Utilisation: ./cuda-copie-dynamique N\n");
    return 0;
  }
  N = atoi(argv[1]);

  // Initialisation
  A = (float *) malloc(sizeof(float) * N);
  B = (float *) malloc(sizeof(float) * N);
  for (i = 0; i < N; i++) { A[i] = (float)i; }
  
  // Allouer le tableau Ad dynamique de taille N sur le GPU avec cudaMalloc 
  hipMalloc(&Ad, sizeof(float) * N);
  // cudaMemcpy de A[N] vers Ad[N]
  hipMemcpy(Ad, A, sizeof(float) * N, hipMemcpyHostToDevice);

  // cudaMemcpy de Ad[N] vers B[N]
  hipMemcpy(B, Ad, sizeof(float) * N, hipMemcpyDeviceToHost);

  // Desaollouer le tableau Ad[N] sur le GPU
  hipFree(Ad);

  // Attendre que les kernels GPUs terminent
  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess) {
    printf("L'execution du kernel a echoue avec le code d'erreur \"%s\".\n", hipGetErrorString(cudaerr));
  }

  // Verifier le resultat
  for (i = 0; i < N; i++) { if (A[i] != B[i]) { break; } }
  if (i < N) { cout << "La copie est incorrecte!\n"; }
  else { cout << "La copie est correcte!\n"; }
  free(A);
  free(B);

  return 0;
}
